#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/transpose_layer.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void TransposeDimensionWithLastDimGPU(const int nthreads,
    const Dtype* input_data, Dtype* output_data, const int count_dim_1, const int count_after_dim_1, 
    const int count_dim_2, const int count_after_dim_2, const int new_count_dim_1,
    const int new_count_after_dim_1, const int new_count_dim_2) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int index_pre=index/count_dim_1;
    int index_dim_1=(index%count_dim_1)/count_after_dim_1;
    int index_insied=(index%count_after_dim_1)/count_dim_2;
    int index_dim_2=(index%count_dim_2)/count_after_dim_2;
    int index_after=index%count_after_dim_2;
    int new_index=index_pre*new_count_dim_1+
                  index_dim_2*new_count_after_dim_1+
                  index_insied*new_count_dim_2+
                  index_dim_1*count_after_dim_2+
                  index_after;
    output_data[new_index]=input_data[index];
  }
}

template <typename Dtype>
__global__ void TransposeDimensionWithoutLastDimGPU(const int nthreads,
    const Dtype* input_data, Dtype* output_data, const int count_dim_1, const int count_after_dim_1, 
    const int count_dim_2, const int new_count_dim_1,
    const int new_count_after_dim_1, const int new_count_dim_2) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int index_pre=index/count_dim_1;
    int index_dim_1=(index%count_dim_1)/count_after_dim_1;
    int index_insied=(index%count_after_dim_1)/count_dim_2;
    int index_dim_2=index%count_dim_2;
    int new_index=index_pre*new_count_dim_1+
                  index_dim_2*new_count_after_dim_1+
                  index_insied*new_count_dim_2+
                  index_dim_1;
    output_data[new_index]=input_data[index];
  }
}

template <typename Dtype>
void TransposeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  
  if(bottom[0]->shape().size()>dim_2+1) {
    TransposeDimensionWithLastDimGPU<Dtype><<<CAFFE_GET_BLOCKS(top[0]->count()), CAFFE_CUDA_NUM_THREADS>>>(
                top[0]->count(), bottom[0]->gpu_data(), top[0]->mutable_gpu_data(), bottom[0]->count(dim_1), 
                bottom[0]->count(dim_1+1), bottom[0]->count(dim_2), bottom[0]->count(dim_2+1), top[0]->count(dim_1), 
                top[0]->count(dim_1+1), top[0]->count(dim_2));
    CUDA_POST_KERNEL_CHECK;
  }
  else {
    TransposeDimensionWithoutLastDimGPU<Dtype><<<CAFFE_GET_BLOCKS(top[0]->count()), CAFFE_CUDA_NUM_THREADS>>>(
                top[0]->count(), bottom[0]->gpu_data(), top[0]->mutable_gpu_data(), bottom[0]->count(dim_1), 
                bottom[0]->count(dim_1+1), bottom[0]->count(dim_2), top[0]->count(dim_1), 
                top[0]->count(dim_1+1), top[0]->count(dim_2));
    CUDA_POST_KERNEL_CHECK;
  }
}

template <typename Dtype>
void TransposeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  if(bottom[0]->shape().size()>dim_2+1) {
    TransposeDimensionWithLastDimGPU<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>(
                bottom[0]->count(), top[0]->gpu_diff(), bottom[0]->mutable_gpu_diff(), top[0]->count(dim_1), 
                top[0]->count(dim_1+1), top[0]->count(dim_2), top[0]->count(dim_2+1), bottom[0]->count(dim_1), 
                bottom[0]->count(dim_1+1), bottom[0]->count(dim_2));
    CUDA_POST_KERNEL_CHECK;
  }
  else {
    TransposeDimensionWithoutLastDimGPU<Dtype><<<CAFFE_GET_BLOCKS(bottom[0]->count()), CAFFE_CUDA_NUM_THREADS>>>(
                bottom[0]->count(), top[0]->gpu_diff(), bottom[0]->mutable_gpu_diff(), top[0]->count(dim_1), 
                top[0]->count(dim_1+1), top[0]->count(dim_2), bottom[0]->count(dim_1), 
                bottom[0]->count(dim_1+1), bottom[0]->count(dim_2));
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(TransposeLayer);
}  // namespace caffe
