#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/triplet_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
    void TripletLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
            const vector<Blob<Dtype>*>& top) {
        int count = bottom[0]->count();
        Dtype dis_anchor2pos;
        Dtype dis_anchor2neg;
        Dtype dis_pos2neg;

        // calc diff for anchor2pos, anchor2neg
        caffe_gpu_sub(count, bottom[0]->gpu_data(),
                bottom[1]->gpu_data(),
                diff_anchor2pos_.mutable_gpu_data());
        caffe_gpu_sub(count, bottom[0]->gpu_data(),
                bottom[2]->gpu_data(),
                diff_anchor2neg_.mutable_gpu_data());
        // share pos diff used for bp
        diff_pos_for_bp_.ShareData(diff_anchor2pos_);
        if (intriplet_mining_) {
            caffe_gpu_sub(count, bottom[1]->gpu_data(),
                    bottom[2]->gpu_data(),
                    diff_pos2neg_.mutable_gpu_data());
        } else {
            // if no in-triplet mining, directly share neg diff used for bp
            diff_neg_for_bp_.ShareData(diff_anchor2neg_);
        }

        Dtype loss = 0;
        for (int v = 0; v < batch_size_; ++v) {
            // calc anchor2pos dis
            caffe_gpu_dot(vec_dimension_,
                    diff_anchor2pos_.gpu_data() + v * vec_dimension_,
                    diff_anchor2pos_.gpu_data() + v * vec_dimension_,
                    &dis_anchor2pos);
            vec_loss_.mutable_cpu_data()[v] = alpha_ + dis_anchor2pos;
            // calc anchor2neg dis
            caffe_gpu_dot(vec_dimension_,
                    diff_anchor2neg_.gpu_data() + v * vec_dimension_,
                    diff_anchor2neg_.gpu_data() + v * vec_dimension_,
                    &dis_anchor2neg);
            if (intriplet_mining_) {
                // calc pos2neg ids
                caffe_gpu_dot(vec_dimension_,
                        diff_pos2neg_.gpu_data() + v * vec_dimension_,
                        diff_pos2neg_.gpu_data() + v * vec_dimension_,
                        &dis_pos2neg);
                // if dis(p - n) < dis(a - n) is found, assign the harder one as negative.
                if (dis_pos2neg < dis_anchor2neg) {
                    // with in-triplet mining, neg diff used for bp should be carefully calculated.
                    caffe_copy(vec_dimension_,
                            diff_pos2neg_.gpu_data() + v * vec_dimension_,
                            diff_anchor2neg_.mutable_gpu_data() + v * vec_dimension_);
                    vec_loss_.mutable_cpu_data()[v] -= dis_pos2neg;
                } else {
                    caffe_copy(vec_dimension_,
                            diff_anchor2neg_.gpu_data() + v * vec_dimension_,
                            diff_neg_for_bp_.mutable_gpu_data() + v * vec_dimension_);
                    vec_loss_.mutable_cpu_data()[v] -= dis_anchor2neg;
                }
            } else {
                vec_loss_.mutable_cpu_data()[v] -= dis_anchor2neg;
            }
            vec_loss_.mutable_cpu_data()[v] = std::max(Dtype(0), vec_loss_.cpu_data()[v]);
            loss += vec_loss_.cpu_data()[v];
        }

        loss /= (batch_size_) * Dtype(2);
        top[0]->mutable_cpu_data()[0] = loss;
        if (top.size() == 2) {
            top[1]->ShareData(vec_loss_);
        }
    }

template <typename Dtype>
    void TripletLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
            const vector<bool>& propagate_down,
            const vector<Blob<Dtype>*>& bottom) {
        const Dtype scale = top[0]->cpu_diff()[0] / bottom[0]->num();
        const int n = bottom[0]->count();


        caffe_gpu_sub(n, diff_pos_for_bp_.gpu_data(), diff_neg_for_bp_.gpu_data(),
                bottom[0]->mutable_gpu_diff());
        caffe_gpu_scal(n, scale, bottom[0]->mutable_gpu_diff());

        caffe_gpu_scale(n, -scale, diff_pos_for_bp_.gpu_data(),
                bottom[1]->mutable_gpu_diff());

        caffe_gpu_scale(n, scale, diff_neg_for_bp_.gpu_data(),
                bottom[2]->mutable_gpu_diff());

        for (int v = 0; v < batch_size_; ++v) {
            if (vec_loss_.cpu_data()[v] == 0) {
                caffe_gpu_set(vec_dimension_, Dtype(0),
                        bottom[0]->mutable_gpu_diff() + v * vec_dimension_);
                caffe_gpu_set(vec_dimension_, Dtype(0),
                        bottom[1]->mutable_gpu_diff() + v * vec_dimension_);
                caffe_gpu_set(vec_dimension_, Dtype(0),
                        bottom[2]->mutable_gpu_diff() + v * vec_dimension_);
            }
        }
    }

INSTANTIATE_LAYER_GPU_FUNCS(TripletLossLayer);

}  // namespace caffe
