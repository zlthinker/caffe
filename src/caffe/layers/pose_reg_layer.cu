#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/pose_reg_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void PoseRegLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  // calc trans loss
  int trans_count = bottom[0]->count();
  caffe_sub(
      trans_count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      trans_diff_.mutable_gpu_data());
  Dtype trans_dot;
  caffe_gpu_dot(trans_count, trans_diff_.gpu_data(), trans_diff_.gpu_data(), &trans_dot);
  Dtype trans_loss = trans_dot / bottom[0]->num() / Dtype(2);
  // calc rot loss
  int rot_count = bottom[2]->count();
  Dtype tmp_dot;
  caffe_gpu_dot(rot_count, bottom[3]->gpu_data(), bottom[3]->gpu_data(), &tmp_dot);
  rot_scale_ = sqrt(tmp_dot);
  caffe_sub(
      rot_count,
      bottom[2]->gpu_data(),
      bottom[3]->gpu_data(),
      rot_diff_.mutable_gpu_data());
  Dtype rot_dot;
  caffe_gpu_dot(rot_count, rot_diff_.gpu_data(), rot_diff_.gpu_data(), &rot_dot);
  Dtype rot_loss = rot_dot / bottom[2]->num() / Dtype(2);
  // calc total loss
  top[0]->mutable_gpu_data()[0] = trans_loss + beta_ / rot_scale_ * rot_loss;
}

template <typename Dtype>
void PoseRegLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  // calc diff for trans
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->gpu_diff()[0] / bottom[i]->num();
      caffe_gpu_axpby(
          bottom[i]->count(),              // count
          alpha,                              // alpha
          trans_diff_.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[i]->mutable_gpu_diff());  // b
    }
  }
  // calc diff for rot
  for (int i = 2; i < 4; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 2) ? 1 : -1;
      const Dtype alpha = (i == 2) ? sign * top[0]->gpu_diff()[0] / bottom[i]->num()
          : sign * top[0]->gpu_diff()[0] / bottom[i]->num() / rot_scale_ * beta_;

      caffe_gpu_axpby(
          bottom[i]->count(),              // count
          alpha,                              // alpha
          rot_diff_.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[i]->mutable_gpu_diff());  // b
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(PoseRegLayer);

}  // namespace caffe
