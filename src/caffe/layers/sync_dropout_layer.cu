#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/sync_dropout_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SyncDropoutForward(const int n, const Dtype* in,
    const unsigned int* mask, const unsigned int threshold, const float scale,
    Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] * (mask[index] > threshold) * scale;
  }
}

template <typename Dtype>
void SyncDropoutLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    const int count = bottom[0]->count();
    unsigned int* mask = NULL;
    if (this->phase_ == TRAIN) {
        mask = static_cast<unsigned int*>(rand_vec_.mutable_gpu_data());
        caffe_gpu_rng_uniform(count, mask);
    }
    for (size_t i = 0; i < bottom.size(); i++) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        Dtype* top_data = top[i]->mutable_gpu_data();
        if (this->phase_ == TRAIN) {
            // set thresholds
            // NOLINT_NEXT_LINE(whitespace/operators)
            SyncDropoutForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
                    count, bottom_data, mask, uint_thres_, scale_, top_data);
            CUDA_POST_KERNEL_CHECK;
        } else {
            caffe_copy(count, bottom_data, top_data);
        }
    }
}

template <typename Dtype>
__global__ void SyncDropoutBackward(const int n, const Dtype* in_diff,
    const unsigned int* mask, const unsigned int threshold, const float scale,
    Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * scale * (mask[index] > threshold);
  }
}

template <typename Dtype>
void SyncDropoutLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
    const int count = bottom[0]->count();
    const unsigned int* mask = NULL;
    if (this->phase_ == TRAIN) {
        mask = static_cast<const unsigned int*>(rand_vec_.gpu_data());
    }
    for (size_t i = 0; i < bottom.size(); i++) {
        if (propagate_down[i]) {
            const Dtype* top_diff = top[i]->gpu_diff();
            Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
            if (this->phase_ == TRAIN) {
                // NOLINT_NEXT_LINE(whitespace/operators)
                SyncDropoutBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
                    CAFFE_CUDA_NUM_THREADS>>>(
                            count, top_diff, mask, uint_thres_, scale_, bottom_diff);
                CUDA_POST_KERNEL_CHECK;
            } else {
                caffe_copy(count, top_diff, bottom_diff);
            }
        }
    }
}

INSTANTIATE_LAYER_GPU_FUNCS(SyncDropoutLayer);

}  // namespace caffe
