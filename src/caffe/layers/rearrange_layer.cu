#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/rearrange_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Rearrange(const int nthreads, const Dtype* in_data,
                          const bool forward, const int group_num, const int group_id,
                          const int channel_num,    //channel num of a single bottom
                          const int dim,    // size of each channel, height*width
                          Dtype* out_data) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        const int channel_id = index / dim;
        const int loc_offset = index % dim;
        const int batch_id = channel_id / channel_num;
        const int channel_offset = channel_id % channel_num;
        const int batch_start = batch_id * (channel_num * dim) * group_num;
        const int channel_start = (group_id + channel_offset * group_num) * dim;
        const int top_index = batch_start + channel_start + loc_offset;

        if (forward) {
            out_data[top_index] = in_data[index];
        } else {
            out_data[index] = in_data[top_index];
        }
    }
}

template <typename Dtype>
void RearrangeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                     const vector<Blob<Dtype>*>& top) {
    Dtype* top_data = top[0]->mutable_gpu_data();
    const bool kForward = true;
    const int group_num = bottom.size();
    const int count = bottom[0]->count();
    const int channel_num = bottom[0]->shape(1);
    const int dim = bottom[0]->shape(2) * bottom[0]->shape(3);
    for (int i = 0; i < bottom.size(); ++i) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        Rearrange<Dtype>
                <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
                    count, bottom_data, kForward, group_num, i,
                    channel_num, dim, top_data);
    }
}

template <typename Dtype>
void RearrangeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    const Dtype* top_diff = top[0]->gpu_diff();
    const bool kForward = false;
    const int group_num = bottom.size();
    const int count = bottom[0]->count();
    const int channel_num = bottom[0]->shape(1);
    const int dim = bottom[0]->shape(2) * bottom[0]->shape(3);
    for (int i = 0; i < bottom.size(); ++i) {
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        Rearrange<Dtype>
                <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
                    count, top_diff, kForward, group_num, i,
                    channel_num, dim, bottom_diff);
    }
}

INSTANTIATE_LAYER_GPU_FUNCS(RearrangeLayer);

}  // namespace caffe
