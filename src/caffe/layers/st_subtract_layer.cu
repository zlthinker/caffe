#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/st_subtract_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
	__global__ void StSubtractForwardGPU(const int nthreads,
			const Dtype* bottom_data0, const Dtype* bottom_data1, Dtype* top_data) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			top_data[index] = bottom_data1[index] - bottom_data0[index];
		}
	}

template <typename Dtype>
void StSubtractLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();
  const Dtype* bottom_data0 = bottom[0]->gpu_data();
  const Dtype* bottom_data1 = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int nthreads = top[0]->count();
  StSubtractForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
		  nthreads, bottom_data0, bottom_data1, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
	__global__ void StSubtractBackwardGPU(const int nthreads,
			Dtype* bottom_diff0, Dtype* bottom_diff1, const Dtype* top_diff) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			bottom_diff0[index] = -top_diff[index];
			bottom_diff1[index] = top_diff[index];
		}
	}

template <typename Dtype>
void StSubtractLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) { return; }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff0 = bottom[0]->mutable_gpu_diff();
  Dtype* bottom_diff1 = bottom[1]->mutable_gpu_diff();

  const int nthreads = top[0]->count();
  StSubtractBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
		  nthreads, bottom_diff0, bottom_diff1, top_diff);
  CUDA_POST_KERNEL_CHECK;
  /*const Dtype* test_bottom_diff0 = bottom[0]->cpu_diff();*/
  /*const Dtype* test_bottom_diff1 = bottom[1]->cpu_diff();*/
  /*for(int i = 0; i < nthreads; i++)*/
	  /*LOG(INFO) << test_bottom_diff0[i] << '\t' << test_bottom_diff1[i];*/
}

INSTANTIATE_LAYER_GPU_FUNCS(StSubtractLayer);

}  // namespace caffe
