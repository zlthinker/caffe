#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/gpu_util.cuh"
#include "caffe/layers/atan2_layer.hpp"
#include "caffe/util/benchmark.hpp"

namespace caffe {

template <typename Dtype>
	__global__ void ATan2ForwardGPU(const int nthreads,
			const Dtype* input, Dtype* output) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			output[index] = atan2(input[2*index], input[2*index + 1]);
		}
	}

template <typename Dtype>
	void ATan2Layer<Dtype>::Forward_gpu(
			const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
		const Dtype* input = bottom[0]->gpu_data();
		Dtype* output = top[0]->mutable_gpu_data();

		CHECK(top[0]->count()*2 == bottom[0]->count()) << "Error: in Forward_gpu of ATan2Layer.";

		const int nthreads = top[0]->count();
		ATan2ForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
				nthreads, input, output);
		CUDA_POST_KERNEL_CHECK;
	}

template <typename Dtype>
	__global__ void ATan2BackwardGPU(const int nthreads, const Dtype* input_data,
			const Dtype* output_diff, Dtype* input_diff) {

		CUDA_KERNEL_LOOP(index, nthreads) {
			Dtype x = input_data[2*index + 1];
			Dtype y = input_data[2*index];
			Dtype eps = 1e-8;
			Dtype deno = x*x + y*y + eps;
			input_diff[2*index] = x/deno * output_diff[index];
			input_diff[2*index+1] = -y/deno * output_diff[index];
		}
	}

template <typename Dtype>
	void ATan2Layer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
			const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

		const Dtype* output_diff = top[0]->gpu_diff();
		const Dtype* input_data = bottom[0]->gpu_data();
		Dtype* input_diff = bottom[0]->mutable_gpu_diff();

		CHECK(top[0]->count()*2 == bottom[0]->count()) << "Error: in Backward_gpu of ATan2Layer.";

		const int nthreads = top[0]->count();
		ATan2BackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
				nthreads, input_data, output_diff, input_diff);
		CUDA_POST_KERNEL_CHECK;
//		const Dtype* test_bottom = bottom[0]->cpu_data();
//		const Dtype* test_top = top[0]->cpu_data();
//		const Dtype* diff_bottom = bottom[0]->cpu_diff();
//		const Dtype* diff_top = top[0]->cpu_diff();
//		for(size_t i = 0; i < top[0]->count(); i++) {
//			LOG(INFO) << "bottom: " << test_bottom[2*i] << '\t' << test_bottom[2*i + 1];
//			LOG(INFO) << "top: " << test_top[i];
//			LOG(INFO) << "diff bottom: " << diff_bottom[2*i] << '\t' << diff_bottom[2*i + 1];
//			LOG(INFO) << "top bottom: " << diff_top[i] << std::endl;
//		}
	}

INSTANTIATE_LAYER_GPU_FUNCS(ATan2Layer);

}	// namespace caffe
