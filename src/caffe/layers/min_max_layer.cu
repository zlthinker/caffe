#include "hip/hip_runtime.h"
#include <vector>
#include <cfloat>

#include "caffe/layers/min_max_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void RescaleForward(const int nthreads,
    const Dtype* const bottom_data,
    const int dim, const Dtype* const min, const Dtype* const gap,
    Dtype* const top_data) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        int channel_id = index / dim;
        Dtype min_val = min[channel_id];
        Dtype gap_val = gap[channel_id];
        top_data[index] = (bottom_data[index] - min_val) / gap_val;
    }
}

template <typename Dtype>
__global__ void RescaleBackward(const int nthreads,
    const Dtype* const top_diff,
    const int dim, const Dtype* const gap,
    Dtype* const bottom_diff) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        int channel_id = index / dim;
        Dtype gap_val = gap[channel_id];
        bottom_diff[index] = top_diff[index] * gap_val;
    }
}



template <typename Dtype>
void MinMaxLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    int num = bottom[0]->num();
    int channel = bottom[0]->channels();
    int dim = bottom[0]->height() * bottom[0]->width();
    int count = bottom[0]->count();

    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    float delta = this->layer_param_.min_max_param().delta();
    CHECK_GT(delta, 0.0);
    CHECK_LT(delta, 1.0);

    shared_ptr<ConstantFiller<Dtype> > filler;
    FillerParameter filler_param;
    filler_param.set_value(FLT_MAX);
    filler.reset(new ConstantFiller<Dtype>(filler_param));
    filler->Fill(&this->min_);
    filler_param.set_value(-FLT_MAX);
    filler.reset(new ConstantFiller<Dtype>(filler_param));
    filler->Fill(&this->max_);
    filler_param.set_value(1.);
    filler.reset(new ConstantFiller<Dtype>(filler_param));
    filler->Fill(&this->gap_);

    // find min and max values within each channel
    for (int n = 0; n < num; n++) {
        for (int c = 0; c < channel; c++) {
            const Dtype* feat_map = bottom[0]->cpu_data() + (n * channel + c) * dim;
            Dtype* min_val = this->min_.mutable_cpu_data() + n * channel + c;
            Dtype* max_val = this->max_.mutable_cpu_data() + n * channel + c;
            Dtype* gap_val = this->gap_.mutable_cpu_data() + n * channel + c;
            for (int d = 0; d < dim; d++) {
                Dtype val = feat_map[d];
                if (val > max_val[0]) { max_val[0] = val; }
                if (val < min_val[0]) { min_val[0] = val; }
            }
            if (max_val[0] - min_val[0] > delta) { gap_val[0] = max_val[0] - min_val[0]; }
//            std::cout << c << ", max: " << max_val[0] << ", min: " << min_val[0]
//                      << ", gap: " << gap_val[0] << "\n";
        }
    }

    // [optional] find min and max values across channels
    if (this->layer_param_.min_max_param().across_channels()) {
        std::cout << "[MinMaxLayer] Rescale activations across channels.\n";
        for (int n = 0; n < num; n++) {
            int start_index = n * channel;
            Dtype min_val = this->min_.cpu_data()[start_index];
            Dtype max_val = this->max_.cpu_data()[start_index];
            for (int c = 0; c < channel; c++) {
                Dtype val1 = this->min_.cpu_data()[start_index + c];
                if (val1 < min_val) { min_val = val1; }
                Dtype val2 = this->max_.cpu_data()[start_index + c];
                if (val2 > max_val) { max_val = val2; }
            }
            Dtype gap_val = (max_val - min_val > delta) ? (max_val - min_val) : 1.0;
            for (int c = 0; c < channel; c++) {
                this->min_.mutable_cpu_data()[start_index + c] = min_val;
                this->max_.mutable_cpu_data()[start_index + c] = max_val;
                this->gap_.mutable_cpu_data()[start_index + c] = gap_val;
            }
            std::cout << n << ", max: " << max_val << ", min: " << min_val
                      << ", gap: " << gap_val << "\n";
        }
    }


    RescaleForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, dim, this->min_.gpu_data(), this->gap_.gpu_data(), top_data);

}

template <typename Dtype>
void MinMaxLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    int dim = bottom[0]->height() * bottom[0]->width();

    RescaleBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, dim, this->gap_.gpu_data(), bottom_diff);
}


INSTANTIATE_LAYER_GPU_FUNCS(MinMaxLayer);


}  // namespace caffe
