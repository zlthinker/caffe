#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/gpu_util.cuh"
#include "caffe/layers/angle_convert_layer.hpp"
#include "caffe/util/benchmark.hpp"

namespace caffe {

template <typename Dtype>
	__global__ void AngleConvertForwardGPU(const int nthreads,
			const Dtype* input, Dtype* output) {

		CUDA_KERNEL_LOOP(index, nthreads) {
			const int offset = index % 6;
			int batchID = index >= 6 ? (index - offset) / 6 : 0;
			Dtype m_cos = cos(input[batchID]);
			Dtype m_sin = sin(input[batchID]);
			switch(offset) {
				case 0:
					output[index] = m_cos;
					break;
				case 1:
					output[index] = m_sin;
					break;
				case 2:
					output[index] = 0;
					break;
				case 3:
					output[index] = -m_sin;
					break;
				case 4:
					output[index] = m_cos;
					break;
				case 5:
					output[index] = 0;
					break;
			}
		}
	}

template <typename Dtype>
	void AngleConvertLayer<Dtype>::Forward_gpu(
			const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
		const Dtype* input = bottom[0]->gpu_data();
		Dtype* output = top[0]->mutable_gpu_data();

		CHECK(top[0]->count() == bottom[0]->count()*6) << "Error: in Forward_gpu of AngleConvertLayer.";

		const int nthreads = top[0]->count();
		AngleConvertForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
				nthreads, input, output);
		CUDA_POST_KERNEL_CHECK;
	}

template <typename Dtype>
	__global__ void AngleConvertBackwardGPU(const int nthreads, const Dtype* output_data,
			const Dtype* output_diff, Dtype* input_diff) {

		CUDA_KERNEL_LOOP(index, nthreads) {
			Dtype m_cos = output_data[index*6 + 0];
			Dtype m_sin = output_data[index*6 + 1];
			input_diff[index] = -m_sin * output_diff[index*6 + 0]
				+ m_cos * output_diff[index*6 + 1]
				- m_cos * output_diff[index*6 + 3]
				- m_sin * output_diff[index*6 + 4];
		}
	}

template <typename Dtype>
	void AngleConvertLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
			const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

		const Dtype* output_diff = top[0]->gpu_diff();
		const Dtype* output_data = top[0]->gpu_data();
		Dtype* input_diff = bottom[0]->mutable_gpu_diff();

		CHECK(top[0]->count() == bottom[0]->count()*6) << "Error: in Backward_gpu of AngleConvertLayer.";

		const int nthreads = bottom[0]->count();
		AngleConvertBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
				nthreads, output_data, output_diff, input_diff);
		CUDA_POST_KERNEL_CHECK;
	}

INSTANTIATE_LAYER_GPU_FUNCS(AngleConvertLayer);

}	// namespace caffe
